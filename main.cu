#include "hip/hip_runtime.h"
#include "common.h"
#include "mmio_highlevel.h"
#include "utils.h"
#include "tranpose.h"
#include "findlevel.h"

#include "sptrsv_syncfree_serialref.h"
#include "sptrsv_syncfree_cuda.cuh"

int main(int argc, char ** argv)
{
    // report precision of floating-point
    printf("---------------------------------------------------------------------------------------------\n");
    char  *precision;
    if (sizeof(VALUE_TYPE) == 4)
    {
        precision = (char *)"32-bit Single Precision";
    }
    else if (sizeof(VALUE_TYPE) == 8)
    {
        precision = (char *)"64-bit Double Precision";
    }
    else
    {
        printf("Wrong precision. Program exit!\n");
        return 0;
    }

    printf("PRECISION = %s\n", precision);
    printf("Benchmark REPEAT = %i\n", BENCH_REPEAT);
    printf("---------------------------------------------------------------------------------------------\n");

    int m, n, nnzA, isSymmetricA;
    int *csrRowPtrA;
    int *csrColIdxA;
    VALUE_TYPE *csrValA;

    int *csrRowPtrTR;
    int *csrColIdxTR;
    VALUE_TYPE *csrValTR;

    int nnzTR;
    int *cscRowIdxTR;
    int *cscColPtrTR;
    VALUE_TYPE *cscValTR;

    int device_id = 0;
    int rhs = 0;
    int substitution = SUBSTITUTION_FORWARD;

    // "Usage: ``./sptrsv -d 0 -rhs 1 -forward -mtx A.mtx'' for LX=B on device 0"
    int argi = 1;

    // load device id
    char *devstr;
    if(argc > argi)
    {
        devstr = argv[argi];
        argi++;
    }

    if (strcmp(devstr, "-d") != 0) return 0;

    if(argc > argi)
    {
        device_id = atoi(argv[argi]);
        argi++;
    }
    printf("device_id = %i\n", device_id);

    // load the number of right-hand-side
    char *rhsstr;
    if(argc > argi)
    {
        rhsstr = argv[argi];
        argi++;
    }

    if (strcmp(rhsstr, "-rhs") != 0) return 0;

    if(argc > argi)
    {
        rhs = atoi(argv[argi]);
        argi++;
    }
    printf("rhs = %i\n", rhs);

    // load substitution, forward or backward
    char *substitutionstr;
    if(argc > argi)
    {
        substitutionstr = argv[argi];
        argi++;
    }

    if (strcmp(substitutionstr, "-forward") == 0)
        substitution = SUBSTITUTION_FORWARD;
    else if (strcmp(substitutionstr, "-backward") == 0)
        substitution = SUBSTITUTION_BACKWARD;
    printf("substitutionstr = %s\n", substitutionstr);
    printf("substitution = %i\n", substitution);

    // load matrix file type, mtx, cscl, or cscu
    char *matstr;
    if(argc > argi)
    {
        matstr = argv[argi];
        argi++;
    }
    printf("matstr = %s\n", matstr);

    // load matrix data from file
    char  *filename;
    if(argc > argi)
    {
        filename = argv[argi];
        argi++;
    }
    printf("-------------- %s --------------\n", filename);

    srand(time(NULL));
    if (strcmp(matstr, "-mtx") == 0)
    {
        // load mtx data to the csr format
        mmio_info(&m, &n, &nnzA, &isSymmetricA, filename);
        csrRowPtrA = (int *)malloc((m+1) * sizeof(int));
        csrColIdxA = (int *)malloc(nnzA * sizeof(int));
        csrValA    = (VALUE_TYPE *)malloc(nnzA * sizeof(VALUE_TYPE));
        mmio_data(csrRowPtrA, csrColIdxA, csrValA, filename);
        printf("input matrix A: ( %i, %i ) nnz = %i\n", m, n, nnzA);

        // extract L or U with a unit diagonal of A
        csrRowPtrTR = (int *)malloc((m+1) * sizeof(int));
        csrColIdxTR = (int *)malloc((m+nnzA) * sizeof(int));
        csrValTR    = (VALUE_TYPE *)malloc((m+nnzA) * sizeof(VALUE_TYPE));

        int nnz_pointer = 0;
        csrRowPtrTR[0] = 0;
        for (int i = 0; i < m; i++)
        {
            for (int j = csrRowPtrA[i]; j < csrRowPtrA[i+1]; j++)
            {   
                if (substitution == SUBSTITUTION_FORWARD)
                {
                    if (csrColIdxA[j] < i)
                    {
                        csrColIdxTR[nnz_pointer] = csrColIdxA[j];
                        //csrValTR[nnz_pointer] = rand() % 10 + 1; 
                        csrValTR[nnz_pointer] = csrValA[j]; 
                        nnz_pointer++;
                    }
                }
                else if (substitution == SUBSTITUTION_BACKWARD)
                {
                    if (csrColIdxA[j] > i)
                    {
                        csrColIdxTR[nnz_pointer] = csrColIdxA[j];
                        //csrValTR[nnz_pointer] = rand() % 10 + 1; 
                        csrValTR[nnz_pointer] = csrValA[j];
                        nnz_pointer++;
                    }
                }
            }

            // add dia nonzero
            csrColIdxTR[nnz_pointer] = i;
            csrValTR[nnz_pointer] = 1.0;
            nnz_pointer++;

            csrRowPtrTR[i+1] = nnz_pointer;
        }

        int nnz_tmp = csrRowPtrTR[m];
        nnzTR = nnz_tmp;

        if (substitution == SUBSTITUTION_FORWARD)
            printf("A's unit-lower triangular L: ( %i, %i ) nnz = %i\n", m, n, nnzTR);
        else if (substitution == SUBSTITUTION_BACKWARD)
            printf("A's unit-upper triangular U: ( %i, %i ) nnz = %i\n", m, n, nnzTR);

        csrColIdxTR = (int *)realloc(csrColIdxTR, sizeof(int) * nnzTR);
        csrValTR = (VALUE_TYPE *)realloc(csrValTR, sizeof(VALUE_TYPE) * nnzTR);

        cscRowIdxTR = (int *)malloc(nnzTR * sizeof(int));
        cscColPtrTR = (int *)malloc((n+1) * sizeof(int));
        memset(cscColPtrTR, 0, (n+1) * sizeof(int));
        cscValTR    = (VALUE_TYPE *)malloc(nnzTR * sizeof(VALUE_TYPE));

        // transpose from csr to csc
        matrix_transposition(m, n, nnzTR,
                             csrRowPtrTR, csrColIdxTR, csrValTR,
                             cscRowIdxTR, cscColPtrTR, cscValTR);

        // keep each column sort 
        for (int i = 0; i < n; i++)
        {
            quick_sort_key_val_pair<int, int>(&cscRowIdxTR[cscColPtrTR[i]],
                                              &cscRowIdxTR[cscColPtrTR[i]],
                                              cscColPtrTR[i+1]-cscColPtrTR[i]);
        }

        // check unit diagonal
        int dia_miss = 0;
        for (int i = 0; i < n; i++)
        {
            bool miss;
            if (substitution == SUBSTITUTION_FORWARD)
                miss = cscRowIdxTR[cscColPtrTR[i]] != i;
            else if (substitution == SUBSTITUTION_BACKWARD)
                cscRowIdxTR[cscColPtrTR[i+1] - 1] != i;

            if (miss) dia_miss++;
        }
        //printf("dia miss = %i\n", dia_miss);
        if (dia_miss != 0) 
        {
            printf("This matrix has incomplete diagonal, #missed dia nnz = %i\n", dia_miss); 
            return;
        }

        free(csrColIdxA);
        free(csrValA);
        free(csrRowPtrA);
    }
    else if (strcmp(matstr, "-csc") == 0)
    {
        FILE *f;
        int returnvalue;

        if ((f = fopen(filename, "r")) == NULL)
            return -1;

        returnvalue = fscanf(f, "%d", &m);
        returnvalue = fscanf(f, "%d", &n);
        returnvalue = fscanf(f, "%d", &nnzTR);

        cscColPtrTR = (int *)malloc((n+1) * sizeof(int));
        memset(cscColPtrTR, 0, (n+1) * sizeof(int));
        cscRowIdxTR = (int *)malloc(nnzTR * sizeof(int));
        cscValTR    = (VALUE_TYPE *)malloc(nnzTR * sizeof(VALUE_TYPE));

        // read row idx
        for (int i = 0; i < n+1; i++)
        {
            returnvalue = fscanf(f, "%d", &cscColPtrTR[i]);
            cscColPtrTR[i]--; // from 1-based to 0-based
        }

        // read col idx
        for (int i = 0; i < nnzTR; i++)
        {
            returnvalue = fscanf(f, "%d", &cscRowIdxTR[i]);
            cscRowIdxTR[i]--; // from 1-based to 0-based
        }

        // read val
        for (int i = 0; i < nnzTR; i++)
        {
            cscValTR[i] = rand() % 10 + 1;
            //returnvalue = fscanf(f, "%lg", &cscValTR[i]);
        }

        if (f != stdin)
            fclose(f);

        // keep each column sort 
        for (int i = 0; i < n; i++)
        {
            quick_sort_key_val_pair<int, int>(&cscRowIdxTR[cscColPtrTR[i]],
                                              &cscRowIdxTR[cscColPtrTR[i]],
                                              cscColPtrTR[i+1]-cscColPtrTR[i]);
        }

        if (substitution == SUBSTITUTION_FORWARD)
            printf("Input csc unit-lower triangular L: ( %i, %i ) nnz = %i\n", m, n, nnzTR);
        else if (substitution == SUBSTITUTION_BACKWARD)
            printf("Input csc unit-upper triangular U: ( %i, %i ) nnz = %i\n", m, n, nnzTR);
       
        // check unit diagonal
        int dia_miss = 0;
        for (int i = 0; i < n; i++)
        {
            bool miss;
            if (substitution == SUBSTITUTION_FORWARD)
                miss = cscRowIdxTR[cscColPtrTR[i]] != i;
            else if (substitution == SUBSTITUTION_BACKWARD)
                cscRowIdxTR[cscColPtrTR[i+1] - 1] != i;

            if (miss) dia_miss++;
        }
        //printf("dia miss = %i\n", dia_miss);
        if (dia_miss != 0) 
        {
            printf("This matrix has incomplete diagonal, #missed dia nnz = %i\n", dia_miss); 
            return;
        }
    }

    // find level sets
    int nlevel = 0;
    int parallelism_min = 0;
    int parallelism_avg = 0;
    int parallelism_max = 0;
    int  *levelPtr  = (int *)malloc((m+1) * sizeof(int));
    int  *levelItem = (int *)malloc(m * sizeof(int));
    findlevel_csc(cscColPtrTR, cscRowIdxTR, cscValTR, m, n, nnzTR, &nlevel,
                  &parallelism_min, &parallelism_avg, &parallelism_max,levelPtr,levelItem);
    double fparallelism = (double)m/(double)nlevel;
    printf("This matrix/graph has %i levels, its parallelism is %4.2f (min: %i ; avg: %i ; max: %i )\n", 
           nlevel, fparallelism, parallelism_min, parallelism_avg, parallelism_max);

    // x and b are all row-major
    VALUE_TYPE *x_ref = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * n * rhs);
    for ( int i = 0; i < n; i++)
        for (int j = 0; j < rhs; j++)
            x_ref[i * rhs + j] = rand() % 10 + 1; //j + 1;

    VALUE_TYPE *b = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * m * rhs);
    VALUE_TYPE *x = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * n * rhs);

    for (int i = 0; i < m * rhs; i++)
        b[i] = 0;

    for (int i = 0; i < n * rhs; i++)
        x[i] = 0;

    // run csc spmv to generate b
    for (int i = 0; i < n; i++)
    {
        for (int j = cscColPtrTR[i]; j < cscColPtrTR[i+1]; j++)
        {
            int rowid = cscRowIdxTR[j]; //printf("rowid = %i\n", rowid);
            for (int k = 0; k < rhs; k++)
            {
                b[rowid * rhs + k] += cscValTR[j] * x_ref[i * rhs + k];
            }
        }
    }

    // run serial syncfree SpTRSV as a reference
    printf("---------------------------------------------------------------------------------------------\n");
    sptrsv_syncfree_serialref(cscColPtrTR, cscRowIdxTR, cscValTR, m, n, nnzTR,
                              substitution, rhs, x, b, x_ref);

    // set device
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    printf("---------------------------------------------------------------------------------------------\n");
    printf("Device [ %i ] %s @ %4.2f MHz\n", device_id, deviceProp.name, deviceProp.clockRate * 1e-3f);

    // run cuda syncfree SpTRSV or SpTRSM

    printf("---------------------------------------------------------------------------------------------\n");
    double gflops_autotuned = 0;
    sptrsvCSC_syncfree_cuda(cscColPtrTR, cscRowIdxTR, cscValTR, m, n, nnzTR,
                         substitution, rhs, OPT_WARP_AUTO, x, b, x_ref, &gflops_autotuned,levelItem);

    printf("---------------------------------------------------------------------------------------------\n");


    printf("---------------------------------------------------------------------------------------------\n");
     gflops_autotuned = 0;
    sptrsvCSR_syncfree_cuda(csrRowPtrTR, csrColIdxTR, csrValTR, m, n, nnzTR,
                         substitution, rhs, OPT_WARP_AUTO, x, b, x_ref, &gflops_autotuned);

    printf("---------------------------------------------------------------------------------------------\n");

    // done!

    free(csrColIdxTR);
    free(csrValTR);
    free(csrRowPtrTR);

    free(cscRowIdxTR);
    free(cscColPtrTR);
    free(cscValTR);

    free(levelPtr);
    free(levelItem);

    free(x);
    free(x_ref);
    free(b);

    return 0;
}
